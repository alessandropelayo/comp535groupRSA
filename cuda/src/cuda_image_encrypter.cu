#include "hip/hip_runtime.h"
#include "image_handler.hpp"
#include "util.hpp"
#include <hip/hip_runtime.h>
#include <iostream>

__device__ long long modExp(long long base, long long exp, long long mod) {
    long long result = 1;
    base = base % mod;
    while (exp > 0) {
        if (exp % 2 == 1)
            result = (result * base) % mod;
        exp = exp >> 1;
        base = (base * base) % mod;
    }
    return result;    
}

__global__ void encrypt_pixel(unsigned char* pixel_data, size_t size, long long public_key, long long n) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        pixel_data[i] = static_cast<unsigned char>(modExp(pixel_data[i], public_key, n));
    }
}

__host__ void setup_kernel(Image &image, long long public_key, long long n) {
    // Length of the image array being processed.
    size_t pixel_data_len = static_cast<size_t>(image.get_height()) 
                          * static_cast<size_t>(image.get_width()) 
                          * static_cast<size_t>(image.get_channels());
                          
    // Size of the image array for memcpy.
    size_t pixel_data_size = pixel_data_len * sizeof(unsigned char);

    // 512 threads per block.
    dim3 block_size (512);

    // Basically just dividing the size of the array against the number of threads per block.
    // The formula is for rounding up in integer division.
    dim3 grid_size ((pixel_data_size + block_size.x - 1) / block_size.x);

    unsigned char* pixel_data;
    hipMalloc(&pixel_data, pixel_data_size);
    hipMemcpy(pixel_data, 
               image.get_pixels(),
               pixel_data_size,
               hipMemcpyHostToDevice);


    verbose("Launching CUDA encryption on file: " + *image.get_filepath());
    encrypt_pixel<<<grid_size, block_size>>>(pixel_data, pixel_data_len, public_key, n);
    hipDeviceSynchronize();

    hipMemcpy(image.get_pixels(), 
            pixel_data,
            pixel_data_size,
            hipMemcpyDeviceToHost);
    hipFree(pixel_data);        
}
