#include "hip/hip_runtime.h"
#include "cuda_image_encrypter.cuh"
#include "image.hpp"

namespace {

__device__ long long mod_exp(long long base, long long exp, const long long mod) {
    long long result = 1;
    base = base % mod;
    while (exp > 0) {
        if (exp % 2 == 1)
            result = (result * base) % mod;
        exp = exp >> 1;
        base = (base * base) % mod;
    }
    return result;
}

__global__ void encrypt_pixel(unsigned char* pixel_data, const size_t size,
                              const long long public_key, const long long n) {
    const size_t i {blockIdx.x * blockDim.x + threadIdx.x};
    if (i < size) {
        pixel_data[i] = static_cast<unsigned char>(mod_exp(pixel_data[i], public_key, n));
    }
}

}

__host__ void gpu_encrypt_image(Image& image, const long long public_key, const long long n) {
    // Length of the image array being processed.
    const size_t pixel_data_len { static_cast<size_t>(image.get_height())
                                * static_cast<size_t>(image.get_width())
                                * static_cast<size_t>(image.get_channels()) };

    // Size of the image array for memcpy.
    const size_t pixel_data_size {pixel_data_len * sizeof(unsigned char)};

    // 512 threads per block
    constexpr dim3 block_size {512};

    // Basically just dividing the size of the array against the number of threads per block.
    // The formula is for rounding up in integer division.
    const dim3 grid_size {static_cast<unsigned int>((pixel_data_len + block_size.x - 1) / block_size.x) };

    unsigned char* cuda_pixel_data;
    hipMalloc(&cuda_pixel_data, pixel_data_size);
    hipMemcpy(cuda_pixel_data,
               image.get_pixels(),
               pixel_data_size,
               hipMemcpyHostToDevice);

    encrypt_pixel<<<grid_size, block_size>>>(cuda_pixel_data, pixel_data_len, public_key, n);
    hipDeviceSynchronize();

    unsigned char* host_pixel_data {static_cast<unsigned char*>(malloc(pixel_data_size))};
    hipMemcpy(host_pixel_data,
               cuda_pixel_data,
               pixel_data_size,
               hipMemcpyDeviceToHost);
    image.set_pixels(host_pixel_data);
    hipFree(cuda_pixel_data);
}
